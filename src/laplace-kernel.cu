#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <laplace-cuda.h>

template <int DIM, class Real> __global__ void LaplaceKernel(Real* U, const Real* Xt, const long* trg_interac_start, const long Nt,
                                                const Real* F, const Real* Xs, const long* src_cnt, const long* src_dsp,
                                                const long* trg_src_lst, const long Ninterac) {
  const long t = blockIdx.x * blockDim.x + threadIdx.x;
  if (t >= Nt || trg_interac_start[t] == -1) return;

  Real Xt_[DIM],  U_ = 0;
  for (long k = 0; k < DIM; k++) Xt_[k] = Xt[t*DIM+k];

  const long interac_start = trg_interac_start[t];
  const long trg_node_idx = trg_src_lst[interac_start*2+0];
  for (long interac_idx = interac_start; interac_idx < Ninterac && trg_src_lst[interac_idx*2+0] == trg_node_idx; interac_idx++) {
    const long src_node_idx = trg_src_lst[interac_idx*2+1];
    const long src_dsp_ = src_dsp[src_node_idx];
    const long Ns = src_cnt[src_node_idx];
    const Real* Xs_ = Xs + src_dsp_*DIM;
    const Real* F_ = F + src_dsp_;

    for (long s = 0; s < Ns; s++) {
      Real R2 = 0;
      for (int k = 0; k < DIM; k++) {
        const Real dX = Xt_[k] - Xs_[s*DIM+k];
        R2 += dX * dX;
      }
      if (R2 > 0) {
        if (DIM == 2) U_ += F_[s] * log(R2) * 0.5;
        if (DIM == 3) U_ += F_[s] * rsqrt(R2);
      }
    }
  }
  U[t] = U_;
}


template <int DIM, class Real> void LaplaceBatchedCUDA(std::vector<Real>& U, const std::vector<Real>& Xt, const std::vector<long>& trg_cnt, const std::vector<long>& trg_dsp,
                                                 const std::vector<Real>& F, const std::vector<Real>& Xs, const std::vector<long>& src_cnt, const std::vector<long>& src_dsp,
                                                 const std::vector<std::pair<long,long>>& trg_src_lst) {
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(LaplaceKernel<DIM),Real>, hipFuncCachePreferL1);

  std::vector<long> trg_interac_start(Xt.size()/DIM);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < trg_cnt.size(); i++) {
    long interac_offset = std::lower_bound(trg_src_lst.begin(), trg_src_lst.end(), std::make_pair(i,(long)0)) - trg_src_lst.begin();
    for (long j = 0; j < trg_cnt[i]; j++) trg_interac_start[trg_dsp[i]+j] = (trg_src_lst[interac_offset].first == i ? interac_offset : -1);
  }

  Real *Xs_, *Xt_, *F_, *U_;
  hipMalloc(&Xs_, Xs.size()*sizeof(Real));
  hipMalloc(&Xt_, Xt.size()*sizeof(Real));
  hipMalloc(&F_, F.size()*sizeof(Real));
  hipMalloc(&U_, U.size()*sizeof(Real));

  long *trg_interac_start_, *trg_src_lst_, *src_cnt_, *src_dsp_;
  hipMalloc(&trg_interac_start_, trg_interac_start.size()*sizeof(long));
  hipMalloc(&trg_src_lst_, trg_src_lst.size()*2*sizeof(long));
  hipMalloc(&src_cnt_, src_cnt.size()*sizeof(long));
  hipMalloc(&src_dsp_, src_dsp.size()*sizeof(long));

  hipMemcpy(Xs_, &(*Xs.begin()), Xs.size()*sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(Xt_, &(*Xt.begin()), Xt.size()*sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(F_, &(*F.begin()), F.size()*sizeof(Real), hipMemcpyHostToDevice);
  hipMemset(U_, 0, U.size()*sizeof(Real));

  hipMemcpy(trg_interac_start_, &(*trg_interac_start.begin()), trg_interac_start.size()*sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(trg_src_lst_, &(*trg_src_lst.begin()), trg_src_lst.size()*2*sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(src_cnt_, &(*src_cnt.begin()), src_cnt.size()*sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(src_dsp_, &(*src_dsp.begin()), src_dsp.size()*sizeof(long), hipMemcpyHostToDevice);

  const int BLOCK_SIZE = 64;
  const int BLOCK_COUNT = (Xt.size()/DIM+BLOCK_SIZE-1)/BLOCK_SIZE;
  LaplaceKernel<DIM><<<BLOCK_COUNT, BLOCK_SIZE>>>(U_, Xt_, trg_interac_start_, Xt.size()/DIM,
                                                  F_, Xs_, src_cnt_, src_dsp_,
                                                  trg_src_lst_, trg_src_lst.size());

  hipMemcpy(&(*U.begin()), U_, U.size()*sizeof(Real), hipMemcpyDeviceToHost);

  hipFree(Xs_);
  hipFree(Xt_);
  hipFree(F_);
  hipFree(U_);

  hipFree(trg_interac_start_);
  hipFree(trg_src_lst_);
  hipFree(src_cnt_);
  hipFree(src_dsp_);

  //hipDeviceSynchronize(); printf("%s\n", hipGetErrorString(hipGetLastError()));
}

void DeviceSynchronizeCUDA() {
  hipDeviceSynchronize();
}

template void LaplaceBatchedCUDA<2,double>(std::vector<double>& U, const std::vector<double>& Xt, const std::vector<long>& trg_cnt, const std::vector<long>& trg_dsp,
                                     const std::vector<double>& F, const std::vector<double>& Xs, const std::vector<long>& src_cnt, const std::vector<long>& src_dsp,
                                     const std::vector<std::pair<long,long>>& trg_src_lst);

template void LaplaceBatchedCUDA<3,double>(std::vector<double>& U, const std::vector<double>& Xt, const std::vector<long>& trg_cnt, const std::vector<long>& trg_dsp,
                                     const std::vector<double>& F, const std::vector<double>& Xs, const std::vector<long>& src_cnt, const std::vector<long>& src_dsp,
                                     const std::vector<std::pair<long,long>>& trg_src_lst);
